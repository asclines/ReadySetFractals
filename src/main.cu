#include "hip/hip_runtime.h"
#include "fractal-generator/generator.h"
#include "settings/settings_loader.h"

#include <unistd.h>
#include <string>
#include <sstream>

using namespace fractal_generator;

//Foward declarations
void GetOptions(
	int argc,
	char **argv,
	GraphSettings *graph_settings_ptr,
	FractalSettings *fractal_settings_ptr,
	ColorSettings *color_settings
	);

int main(int argc, char **argv){

	fractal_generator::GraphSettings graph_settings;	
        fractal_generator::FractalSettings fractal_settings;
        fractal_generator::ColorSettings color_settings;
        fractal_generator::SetPixelsResults *results_ptr;

/*	
	graph_settings.radius = 1;
	graph_settings.x_offset = 0.0;
	graph_settings.y_offset = 0.0;

	fractal_settings.graph_settings = graph_settings;
	fractal_settings.type = GetFractalTypeFromValue(1);
	fractal_settings.complex_num = complex(-0.45,0.6);
	fractal_settings.dimm = 1024;
	fractal_settings.escape_value = 2;
	fractal_settings.max_iterations = 100;

	color_settings.is_bw = false;
	color_settings.color_option = 100;
*/

/*
	SettingsLoaderError settings_loader_error = LoadSettingsFromSettingsFile(
							&fractal_settings,
							&color_settings
							);

	if(settings_loader_error != OKAY){
		std::cout << "Settings error: " << settings_loader_error_strings[settings_loader_error] << std::endl;
		return settings_loader_error;
	}

*/

	GetOptions(argc,argv,&graph_settings,&fractal_settings,&color_settings);

	 std::cout << "Settings" << std::endl
                << "\tRadius: " << fractal_settings.graph_settings.radius << std::endl
		<< "\tIterations: " << fractal_settings.max_iterations << std::endl;




	results_ptr = fractal_generator::GenerateFractal(fractal_settings);

	std::string file_name = fractal_generator::GenerateImage(color_settings,fractal_settings, results_ptr);

	return 0;
}


void GetOptions(
	int argc,
	char **argv,
        GraphSettings *graph_settings_ptr,
        FractalSettings *fractal_settings_ptr,
        ColorSettings *color_settings
        ){

        extern char *optarg;
        extern int optind;

	int error=0;
	int option;
	
	std::stringstream stream;
		
	int opts_int_holder;

        /*
                f-fractal type
                d-dimm
                e-escape value
                m-max iterations
        */
        while((option = getopt(argc,argv,"f:d:e:m:")) != -1){
                switch(option){
                        case 'f': //Fractal Type
        			stream << optarg;
				stream >> opts_int_holder;
	                        fractal_settings_ptr->type = GetFractalTypeFromValue(opts_int_holder);
				stream.clear();
				break;
			case 'd': //Dimmensions for square
				stream << optarg;
				stream >> opts_int_holder;
				fractal_settings_ptr->dimm = opts_int_holder;
				stream.clear();
				break;
			case 'e': //Escape range
				stream << optarg;
				stream >> opts_int_holder;
				fractal_settings_ptr->escape_value = opts_int_holder;
				stream.clear();
				break;	
			case 'm': //Max iterations
				stream << optarg;
				stream >> opts_int_holder;
				fractal_settings_ptr->max_iterations = opts_int_holder;
				stream.clear();
				break;
			case '?':
				error = 1;
				break;

                }

        }

	if(error >0){
		//TODO Print usage
	}


}
