#include "hip/hip_runtime.h"
#include "fractal-generator/generator.h"
#include "settings/settings_loader.h"

#include <unistd.h>
#include <string>
#include <sstream>
#include <iostream>

using namespace fractal_generator;

//Foward declarations
void GetOptions(
	int argc,
	char **argv,
	GraphSettings *graph_settings_ptr,
	FractalSettings *fractal_settings_ptr,
	ColorSettings *color_settings
	);

int main(int argc, char **argv){

	fractal_generator::GraphSettings graph_settings;	
        fractal_generator::FractalSettings fractal_settings;
        fractal_generator::ColorSettings color_settings;
        fractal_generator::SetPixelsResults *results_ptr;
/*
	SettingsLoaderError settings_loader_error = LoadSettingsFromSettingsFile(
							&fractal_settings,
							&color_settings
							);

	if(settings_loader_error != OKAY){
		std::cout << "Settings error: " << settings_loader_error_strings[settings_loader_error] << std::endl;
		return settings_loader_error;
	}

*/

	GetOptions(argc,argv,&graph_settings,&fractal_settings,&color_settings);
	fractal_settings.graph_settings = graph_settings;

	results_ptr = fractal_generator::GenerateFractal(fractal_settings);

	std::string file_name = fractal_generator::GenerateImage(color_settings,fractal_settings, results_ptr);

	return 0;
}


void GetOptions(
	int argc,
	char **argv,
        GraphSettings *graph_settings_ptr,
        FractalSettings *fractal_settings_ptr,
        ColorSettings *color_settings_ptr
        ){

        extern char *optarg;
        extern int optind;

	int error=0;
	int option;
		
	double constant_imag = -0.65;        	
	double constant_real = 0.45;

        /*
                f-fractal type
                d-dimm
                e-escape value
                m-max iterations
		r-radius
		x-x offset
		y-y offset
		c-color option/seed
		I-imagine
		R-real
        */
        while((option = getopt(argc,argv,"f:d:e:m:r:x:y:c:I:R:")) != -1){
		std::stringstream stream;
		double opts_double_holder;
		int opts_int_holder;

		stream << optarg;
		switch(option){
                        case 'f': //Fractal Type
				stream >> opts_int_holder;
	                        fractal_settings_ptr->type = GetFractalTypeFromValue(opts_int_holder);
				break;
			case 'd': //Dimmensions for square
				stream >> opts_int_holder;
				fractal_settings_ptr->dimm = opts_int_holder;
				break;
			case 'e': //Escape range
				stream >> opts_int_holder;
				fractal_settings_ptr->escape_value = opts_int_holder;
				break;	
			case 'm': //Max iterations
				stream >> opts_int_holder;
				fractal_settings_ptr->max_iterations = opts_int_holder;
				break;
			case 'r': //Radius			
				stream >> opts_double_holder;
				graph_settings_ptr->radius = opts_double_holder;
				break;
			case 'x': //X Offset
				stream >> opts_double_holder;
				graph_settings_ptr->x_offset = opts_double_holder;
				break;
			case 'y': //Y Offset
				stream >> opts_double_holder;
				graph_settings_ptr->y_offset = opts_double_holder;
				break;
			case 'c': //Color seed
				stream >> opts_int_holder;
				color_settings_ptr->color_option = opts_int_holder;
				break;
			case 'I': //Image number
				stream >> constant_imag;
				break;
			case 'R': //Real number
				stream >> constant_real;
				break;
			case '?':
				error = 1;
				break;

                }
		stream.clear();

        }

	fractal_settings_ptr->complex_num = complex(constant_real,constant_imag);
	if(error >0){
		std::cout << "usage: " << argv[0] << " [-f type] [-d dimm] [-e escape] [-m iterations] [-r radius] [-x x_offset] [-y y_offset] [-c color] [-I imaginary] [-R real] " << std::endl;
		exit(1);
	}


}
