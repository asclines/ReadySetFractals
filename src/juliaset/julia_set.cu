#include "julia_set.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <iostream>
#include <string>
#include <qdbmp/qdbmp.h>

#include <colors/color_gen.h>

/* Class JuliaSet member function defintions */
int JuliaSet::GetGPUInfo(){
	int devices_count;
	
	hipGetDeviceCount(&devices_count);
	printf("CUDA Device count: %d\n",devices_count);
	for(int i=0; i<devices_count;i++){
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop,i);
		printf("Device Number: %d\n",i);
		printf("\tDevice Name: %s\n",prop.name);
	}
	return devices_count;
}

bool JuliaSet::GeneratePixels(){
	PrepareSettings();
	if(current_state_ != READY){
		std::cout << "JuliaSet::GeneratePixels() : current_state_ is NOT READY " << std::endl;
		exit(1);
	}
	julia_set_kernel::PrintJuliaSettings(settings_);

	typedef julia_set_kernel::var var;
	typedef julia_set_kernel::Complex Complex;

/* Setting up device and host  structures */
	julia_set_kernel::SetPixelsResults *d_set_pixels_results_ptr;
	var *d_x_pixels_ptr, *d_y_pixels_ptr, *d_escape_iterations_ptr;
	var *h_x_pixels_ptr, *h_y_pixels_ptr, *h_escape_iterations_ptr;

	//Allocate memory
	var var_array_size = settings_.w_pixels * settings_.h_pixels * sizeof(var);
	hipMalloc((void**)&d_set_pixels_results_ptr,sizeof(julia_set_kernel::SetPixelsResults));
	hipMalloc((void**)&d_x_pixels_ptr,var_array_size);
	hipMalloc((void**)&d_y_pixels_ptr,var_array_size);
	hipMalloc((void**)&d_escape_iterations_ptr,var_array_size);

	h_set_pixels_results_ptr_ = (julia_set_kernel::SetPixelsResults*)malloc(sizeof(julia_set_kernel::SetPixelsResults));
	h_x_pixels_ptr = (var*)malloc(var_array_size);
	h_y_pixels_ptr = (var*)malloc(var_array_size);
	h_escape_iterations_ptr = (var*)malloc(var_array_size);

	//Set host pointers to device pointers for memcp
	h_set_pixels_results_ptr_->x_pixels_ptr = d_x_pixels_ptr;
	h_set_pixels_results_ptr_->y_pixels_ptr = d_y_pixels_ptr;
	h_set_pixels_results_ptr_->escape_iterations_ptr = d_escape_iterations_ptr;

	//Copy everything over
	hipMemcpy(d_set_pixels_results_ptr,h_set_pixels_results_ptr_,sizeof(julia_set_kernel::SetPixelsResults),hipMemcpyHostToDevice);

	/*Dealing with kernel*/
	//Setting up dim3 for kernel call
	dim3 threads,blocks;
	threads.x=16;//In future, this needs to not be hardcoded
	threads.y=16;
	blocks.x=(settings_.w_pixels/threads.x);
	blocks.y=(settings_.h_pixels/threads.y);
	
	printf("Starting kernel\n");

	//Calling kernel
	julia_set_kernel::SetPixels<<<blocks,threads>>>(settings_,d_set_pixels_results_ptr);

	//Waiting for kernel to finish	
	hipDeviceSynchronize();
	printf("Kernel done\n");


	//Getting data back
	hipMemcpy(h_set_pixels_results_ptr_,d_set_pixels_results_ptr,sizeof(julia_set_kernel::SetPixelsResults),hipMemcpyDeviceToHost);
	hipMemcpy(h_x_pixels_ptr,d_x_pixels_ptr,var_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(h_y_pixels_ptr,d_y_pixels_ptr,var_array_size,hipMemcpyDeviceToHost);
	hipMemcpy(h_escape_iterations_ptr,d_escape_iterations_ptr,var_array_size,hipMemcpyDeviceToHost);

	//Reassign host pointers
	h_set_pixels_results_ptr_->x_pixels_ptr = h_x_pixels_ptr;
	h_set_pixels_results_ptr_->y_pixels_ptr = h_y_pixels_ptr;
	h_set_pixels_results_ptr_->escape_iterations_ptr = h_escape_iterations_ptr;

	//Free memory
	hipFree(d_set_pixels_results_ptr);
	hipFree(d_x_pixels_ptr);
	hipFree(d_y_pixels_ptr);
	hipFree(d_escape_iterations_ptr);

	current_state_ = GENERATED;	
	return true;
}


bool JuliaSet::GenerateColorImage(int color_offset, const char* image_file_name){
	return GenerateImage(true,color_offset,image_file_name);
}

bool JuliaSet::GenerateBWImage(const char* image_file_name){
	return GenerateImage(false,0,image_file_name);
}

bool JuliaSet::GenerateImage(bool in_color, int color_offset, const char* image_file_name){
	if(current_state_ != GENERATED){
		std::cout << "JuliaSet::GenerateImage() : current_state_ is NOT GENERATED" << std::endl;
		exit(1);
	}

	BMP* bmp;
	int max_colors = get_color_list_size();

	bmp = BMP_Create(settings_.w_pixels,settings_.h_pixels,8);

	//Setting pixel index //In future, this should be moved onto GPU
	for(int i = 0; i < settings_.w_pixels * settings_.h_pixels; i++){
		BMP_SetPixelIndex(
			bmp,
			h_set_pixels_results_ptr_->x_pixels_ptr[i],
			h_set_pixels_results_ptr_->y_pixels_ptr[i],
			h_set_pixels_results_ptr_->escape_iterations_ptr[i]
		);
	}

	//Setting color palette
	BMP_SetPaletteColor(bmp,1,0,0,0);
	for(int i = 1; i < settings_.max_iterations; i++){
		if(in_color){
			RGBColor color = get_rgb_color((i+color_offset)%max_colors);
			BMP_SetPaletteColor(bmp,i,color.red,color.green,color.blue);
		} else{ //Not in color
			BMP_SetPaletteColor(bmp,i,i,i,i);
		}
	}

	BMP_WriteFile(bmp,image_file_name);
	BMP_Free(bmp);
	BMP_CHECK_ERROR(stderr,-2);

return true;
}

bool JuliaSet::PrepareSettings(){
	settings_.w_pixels = w_pixels_;
	settings_.h_pixels = h_pixels_;
	settings_.xy_radius = xy_radius_;
	settings_.x_offset = x_offset_;
	settings_.y_offset = y_offset_;
	settings_.max_iterations = max_iterations_;
	settings_.escape_range = escape_range_;
	settings_.complex_constant = make_hipDoubleComplex(julia_constant_real_,julia_constant_imag_);

	current_state_ = READY;
	return true;
}

/* Getters */

int JuliaSet::get_w_pixels(){
	return w_pixels_;
}


int JuliaSet::get_h_pixels(){
	return h_pixels_;
}

double JuliaSet::get_xy_radius(){
	return xy_radius_;
}

double JuliaSet::get_x_offset(){
	return x_offset_;
}

double JuliaSet::get_y_offset(){
	return y_offset_;
}

int JuliaSet::get_max_iterations(){
	return max_iterations_;
}

int JuliaSet::get_escape_range(){
	return escape_range_;
}


double JuliaSet::get_julia_constant_real(){
	return julia_constant_real_;
}

double JuliaSet::get_julia_constant_imag(){
	return julia_constant_imag_;
}


/* Setters */

void JuliaSet::set_pixels(int w_pixels,int h_pixels){
	w_pixels_ = w_pixels;
	h_pixels_ = h_pixels;
}

void JuliaSet::set_radius(double xy_radius){
	xy_radius_ = xy_radius;
}

void JuliaSet::set_offset(double x_offset, double y_offset){
	x_offset_ = x_offset;
	y_offset_ = y_offset;
}

void JuliaSet::set_max_iterations(int max_iterations){
	max_iterations_ = max_iterations;
}

void JuliaSet::set_escape_range(int escape_range){
	escape_range_ = escape_range;
}

void JuliaSet::set_julia_constant(double julia_constant_real,double julia_constant_imag){
	julia_constant_real_ = julia_constant_real;
	julia_constant_imag_ = julia_constant_imag;
}
