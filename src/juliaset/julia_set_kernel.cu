#include "hip/hip_runtime.h"
#include "julia_set_kernel.cuh"
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

namespace julia_set_kernel{

__global__ void SetPixels(JuliaSettings settings, SetPixelsResults *results_ptr){

/* Setting up required variables */
	//Getting thread id
	var thread_id = ( blockIdx.y*gridDim.x+blockIdx.x)*(blockDim.x*blockDim.y)+(threadIdx.y*blockDim.x)+threadIdx.x;

	//Image variables xPixel/yPixel is the x/y coordinate for the current pixel
	var x_pixel = ((blockIdx.x * blockDim.x) + threadIdx.x);
	var y_pixel = (blockIdx.y * blockDim.y) + threadIdx.y;
		
	//Julia calculation variables
	Complex complex_num;
	int depth = 0;
	

/* Performing all calculations */
	if((x_pixel<settings.w_pixels)&&(y_pixel<settings.h_pixels)){

		results_ptr->x_pixels_ptr[thread_id] = x_pixel;
		results_ptr->y_pixels_ptr[thread_id] = y_pixel;

		//Graph variables
		double x_point,y_point; //To be set later

		// Get Points from pixels 
		x_point = (double)(x_pixel*settings.xy_radius*2)/settings.w_pixels;
		y_point = (double)(y_pixel*settings.xy_radius*2)/settings.h_pixels;		

		// Transform points 
		x_point = x_point - settings.xy_radius + settings.x_offset;
		y_point = y_point - settings.xy_radius + settings.y_offset;

		// Julia Calculations 
		complex_num = make_hipDoubleComplex(x_point,y_point);

		while((hipCabs(complex_num) <= settings.escape_range) && (depth <= settings.max_iterations)){
			depth++;
			complex_num = hipCmul(complex_num,complex_num);
			complex_num = hipCadd(complex_num,settings.complex_constant);
		} 

		// Set values in devicePackage 
		if(hipCabs(complex_num) <=settings.escape_range){
			results_ptr->escape_iterations_ptr[thread_id] = 0;
		} else{
			results_ptr->escape_iterations_ptr[thread_id] = depth;	
		}

	}
}


void PrintJuliaSettings(JuliaSettings settings){
	using namespace std;
	cout << "Printing Julia Settings" << endl;
	cout << "\tImage Size Settings" << endl;
	cout << "\t\tWidth(Pixels): " << settings.w_pixels << endl;
	cout << "\t\tHeight(Pixels): " << settings.h_pixels << endl;
	cout << "\t\tRadius(Points): " << settings.xy_radius << endl;
	cout << "\t\tX Direction Offset(Points): " << settings.x_offset << endl;
	cout << "\t\tY Direction Offset(Points): " << settings.y_offset << endl;
	cout << "\tJulia Fractal Settings" << endl;
	cout << "\t\tMax Iterations: " << settings.max_iterations << endl;
	cout << "\t\tEscape Range(Point): " << settings.escape_range << endl;
	cout << "\t\tJulia Constant Real: " << hipCreal(settings.complex_constant) << endl;
	cout << "\t\tJulia Constant Imag: " << hipCimag(settings.complex_constant) << endl;

}		

} //End namespace
	
