#include "generator.h"
#include "kernel.h"
#include <qdbmp/qdbmp.h>
#include <colors/color_gen.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>

namespace fractal_generator{

SetPixelsResults* GenerateFractal(FractalSettings settings){


/* Setting up device and host  structures */
	SetPixelsResults *d_set_pixels_results_ptr, *h_set_pixels_results_ptr;
        uint *d_x_pixels_ptr, *d_y_pixels_ptr, *d_escape_iterations_ptr;
        uint *h_x_pixels_ptr, *h_y_pixels_ptr, *h_escape_iterations_ptr;

	//Allocate memory
        uint uint_array_size = settings.dimm * settings.dimm * sizeof(uint);
        hipMalloc((void**)&d_set_pixels_results_ptr,sizeof(SetPixelsResults));
        hipMalloc((void**)&d_x_pixels_ptr,uint_array_size);
        hipMalloc((void**)&d_y_pixels_ptr,uint_array_size);
        hipMalloc((void**)&d_escape_iterations_ptr,uint_array_size);

        h_set_pixels_results_ptr = (SetPixelsResults *)malloc(sizeof(SetPixelsResults));
        h_x_pixels_ptr = (uint*)malloc(uint_array_size);
        h_y_pixels_ptr = (uint*)malloc(uint_array_size);
        h_escape_iterations_ptr = (uint*)malloc(uint_array_size);

        //Set host pointers to device pointers for memcp
        h_set_pixels_results_ptr->x_pixels_ptr = d_x_pixels_ptr;
        h_set_pixels_results_ptr->y_pixels_ptr = d_y_pixels_ptr;
        h_set_pixels_results_ptr->escape_iterations_ptr = d_escape_iterations_ptr;

        //Copy everything over
        hipMemcpy(d_set_pixels_results_ptr,h_set_pixels_results_ptr,sizeof(SetPixelsResults),hipMemcpyHostToDevice);

/*Dealing with kernel*/
        //Setting up dim3 for kernel call
        dim3 threads,blocks;
        threads.x=8;//In future, this needs to not be hardcoded
        threads.y=8;
        blocks.x=(settings.dimm/threads.x);
        blocks.y=(settings.dimm/threads.y);

        printf("Starting kernel\n");

        //Calling kernel
	switch(settings.type){
		case FRACTAL::JULIA:
			 SetPixels<FRACTAL::JULIA><<<blocks,threads>>>(settings,d_set_pixels_results_ptr);
		break;
		case FRACTAL::MANDLEBROT:
			SetPixels<FRACTAL::MANDLEBROT><<<blocks,threads>>>(settings,d_set_pixels_results_ptr);
                break;
	}

        //Waiting for kernel to finish  
        hipDeviceSynchronize();
        printf("Kernel done\n");

        //Getting data back
        hipMemcpy(h_set_pixels_results_ptr,d_set_pixels_results_ptr,sizeof(SetPixelsResults),hipMemcpyDeviceToHost);
        hipMemcpy(h_x_pixels_ptr,d_x_pixels_ptr,uint_array_size,hipMemcpyDeviceToHost);
        hipMemcpy(h_y_pixels_ptr,d_y_pixels_ptr,uint_array_size,hipMemcpyDeviceToHost);
        hipMemcpy(h_escape_iterations_ptr,d_escape_iterations_ptr,uint_array_size,hipMemcpyDeviceToHost);

	//Reassign host pointers
        h_set_pixels_results_ptr->x_pixels_ptr = h_x_pixels_ptr;
        h_set_pixels_results_ptr->y_pixels_ptr = h_y_pixels_ptr;
        h_set_pixels_results_ptr->escape_iterations_ptr = h_escape_iterations_ptr;

        //Free memory
        hipFree(d_set_pixels_results_ptr);
        hipFree(d_x_pixels_ptr);
        hipFree(d_y_pixels_ptr);
        hipFree(d_escape_iterations_ptr);

	return h_set_pixels_results_ptr;
}

std::string GenerateImage(ColorSettings color_settings, FractalSettings fractal_settings, SetPixelsResults *results_ptr){
	BMP* bmp;
        int max_colors = color_gen::get_color_list_size();

	char* image_file_name = "output_image.bmp";	

	bmp = BMP_Create(fractal_settings.dimm,fractal_settings.dimm,8);

	printf("Generating image\n");

        //Setting pixel index //In future, this should be moved onto GPU
	for(int i=0; i<fractal_settings.dimm * fractal_settings.dimm; i++){
		BMP_SetPixelIndex(
			bmp,
			results_ptr->x_pixels_ptr[i],
			results_ptr->y_pixels_ptr[i],
			results_ptr->escape_iterations_ptr[i]
		);

	}
	
        //Setting color palette
	//TODO take color palette setting from user instead of hardcod	
	color_gen::Set_Color_Palette(bmp,fractal_settings.max_iterations,2);


        BMP_WriteFile(bmp,image_file_name);
        BMP_Free(bmp);
       // BMP_CHECK_ERROR(stderr,-2);
	
	
	return "end"; //image_file_name;
}

FRACTAL GetFractalTypeFromValue(int value){
	
	switch(value){
		case 1:
			return FRACTAL::JULIA;
		case 2:
			return FRACTAL::MANDLEBROT;
		default:
			return FRACTAL::ERROR;
	}
}


}//End namepsace
