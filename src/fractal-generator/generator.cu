#include "generator.h"
#include "kernel.h"
#include <qdbmp/qdbmp.h>
#include <colors/color_gen.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>

namespace fractal_generator{

SetPixelsResults* GenerateFractal(FractalSettings settings){


/* Setting up device and host  structures */
	SetPixelsResults *d_set_pixels_results_ptr, *h_set_pixels_results_ptr;
        uint *d_x_pixels_ptr, *d_y_pixels_ptr, *d_escape_iterations_ptr;
        uint *h_x_pixels_ptr, *h_y_pixels_ptr, *h_escape_iterations_ptr;

	//Allocate memory
        uint uint_array_size = settings.dimm * settings.dimm * sizeof(uint);
        hipMalloc((void**)&d_set_pixels_results_ptr,sizeof(SetPixelsResults));
        hipMalloc((void**)&d_x_pixels_ptr,uint_array_size);
        hipMalloc((void**)&d_y_pixels_ptr,uint_array_size);
        hipMalloc((void**)&d_escape_iterations_ptr,uint_array_size);

        h_set_pixels_results_ptr = (SetPixelsResults *)malloc(sizeof(SetPixelsResults));
        h_x_pixels_ptr = (uint*)malloc(uint_array_size);
        h_y_pixels_ptr = (uint*)malloc(uint_array_size);
        h_escape_iterations_ptr = (uint*)malloc(uint_array_size);

        //Set host pointers to device pointers for memcp
        h_set_pixels_results_ptr->x_pixels_ptr = d_x_pixels_ptr;
        h_set_pixels_results_ptr->y_pixels_ptr = d_y_pixels_ptr;
        h_set_pixels_results_ptr->escape_iterations_ptr = d_escape_iterations_ptr;

        //Copy everything over
        hipMemcpy(d_set_pixels_results_ptr,h_set_pixels_results_ptr,sizeof(SetPixelsResults),hipMemcpyHostToDevice);

/*Dealing with kernel*/
        //Setting up dim3 for kernel call
        dim3 threads,blocks;
        threads.x=8;//In future, this needs to not be hardcoded
        threads.y=8;
        blocks.x=(settings.dimm/threads.x);
        blocks.y=(settings.dimm/threads.y);

        printf("Starting kernel\n");

        //Calling kernel
        SetPixels<FRACTAL::JULIA><<<blocks,threads>>>(settings,d_set_pixels_results_ptr);

        //Waiting for kernel to finish  
        hipDeviceSynchronize();
        printf("Kernel done\n");

        //Getting data back
        hipMemcpy(h_set_pixels_results_ptr,d_set_pixels_results_ptr,sizeof(SetPixelsResults),hipMemcpyDeviceToHost);
        hipMemcpy(h_x_pixels_ptr,d_x_pixels_ptr,uint_array_size,hipMemcpyDeviceToHost);
        hipMemcpy(h_y_pixels_ptr,d_y_pixels_ptr,uint_array_size,hipMemcpyDeviceToHost);
        hipMemcpy(h_escape_iterations_ptr,d_escape_iterations_ptr,uint_array_size,hipMemcpyDeviceToHost);

	//Reassign host pointers
        h_set_pixels_results_ptr->x_pixels_ptr = h_x_pixels_ptr;
        h_set_pixels_results_ptr->y_pixels_ptr = h_y_pixels_ptr;
        h_set_pixels_results_ptr->escape_iterations_ptr = h_escape_iterations_ptr;

        //Free memory
        hipFree(d_set_pixels_results_ptr);
        hipFree(d_x_pixels_ptr);
        hipFree(d_y_pixels_ptr);
        hipFree(d_escape_iterations_ptr);

	std::cout << "X1" <<h_set_pixels_results_ptr->escape_iterations_ptr[5] << std::endl;
	
	return h_set_pixels_results_ptr;
}

std::string GenerateImage(ColorSettings color_settings, FractalSettings fractal_settings, SetPixelsResults *results_ptr){
	BMP* bmp;
        int max_colors = get_color_list_size();

	char* image_file_name = "output_image.bmp";	

	bmp = BMP_Create(fractal_settings.dimm,fractal_settings.dimm,8);

	printf("Generating image\n");


        //Setting pixel index //In future, this should be moved onto GPU
	for(int i=0; i<fractal_settings.dimm * fractal_settings.dimm; i++){
		printf("%d\n",i);
		BMP_SetPixelIndex(
			bmp,
			results_ptr->x_pixels_ptr[i],
			results_ptr->y_pixels_ptr[i],
			results_ptr->escape_iterations_ptr[i]
		);

	}
	
        //Setting color palette
        BMP_SetPaletteColor(bmp,1,0,0,0);
        for(int i = 1; i < fractal_settings.max_iterations; i++){
                if(!color_settings.is_bw){
                        RGBColor color = get_rgb_color((i+color_settings.color_offset)%max_colors);
                        BMP_SetPaletteColor(bmp,i,color.red,color.green,color.blue);
                } else{ //Not in color
                        BMP_SetPaletteColor(bmp,i,i,i,i);
                }
        }

        BMP_WriteFile(bmp,image_file_name);
        BMP_Free(bmp);
       // BMP_CHECK_ERROR(stderr,-2);
	
	
	return "end"; //image_file_name;
}

}//End namepsace
