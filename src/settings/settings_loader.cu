#include "hip/hip_runtime.h"
#include "settings_loader.h"
#include <fractal-generator/generator.h>

#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
#include <vector>

namespace fractal_generator{

/* Structs & Enums */
struct Settings{
	int dimm;
	double radius_xy;
	double x_offset;
	double y_offset;
	int iterations;
	int escape_range;
	double constant_real;
	double constant_imag;
	FRACTAL type;
	
	int color_offset;
	bool image_in_bw;
};



/* Foward Declarations For Private Methods */

SettingsLoaderError SetFractalSettings(
	const Settings settings,
	FractalSettings *fractal_settings_ptr,
	ColorSettings *color_settings_ptr );

SettingsLoaderError ExtractSettingsFromVector(std::vector<std::string> settings_list, Settings *settings_ptr);

/* Method Definitions */

SettingsLoaderError LoadSettingsFromSettingsFile(
			FractalSettings *fractal_settings_ptr,
                        ColorSettings *color_settings_ptr
                        ){
	
	return LoadSettingsFromFile("settings.txt",fractal_settings_ptr,color_settings_ptr);

}



SettingsLoaderError LoadSettingsFromFile(
			std::string file_name,
                        FractalSettings *fractal_settings_ptr,
                        ColorSettings *color_settings_ptr
                        ){
	std::vector<std::string> settings_list; //All even indices are the setting key and the following odd index is the setting value
	std::ifstream settings_file(file_name.c_str());
	if(settings_file.is_open()){
		std::string settings_line;
		while(std::getline(settings_file,settings_line)){ 
			std::stringstream stream(settings_line);
			std::string settings_item;
			while(getline(stream,settings_item,'=')){
				settings_list.push_back(settings_item);
			}						
		}
		settings_file.close();

		Settings settings;
		SettingsLoaderError error = ExtractSettingsFromVector(settings_list, &settings);
		if(error == OKAY){
			return SetFractalSettings(settings, fractal_settings_ptr, color_settings_ptr);
		} else{
			return error;
		}
	} else{
		std::cout << "Cannot open file " << file_name << std::endl;
		return COULD_NOT_OPEN_FILE;
	}
	return OKAY;
}


SettingsLoaderError SetFractalSettings(
const Settings settings, 
        FractalSettings *fractal_settings_ptr, 
        ColorSettings *color_settings_ptr ){

	fractal_settings_ptr->graph_settings.radius = settings.radius_xy;
	fractal_settings_ptr->graph_settings.x_offset = settings.x_offset;
	fractal_settings_ptr->graph_settings.y_offset = settings.y_offset;

	fractal_settings_ptr->type = settings.type;
	fractal_settings_ptr->complex_num = complex(settings.constant_real,settings.constant_imag);
	fractal_settings_ptr->dimm = settings.dimm;
	fractal_settings_ptr->escape_value = settings.escape_range;
	fractal_settings_ptr->max_iterations = settings.iterations;

	color_settings_ptr->is_bw = settings.image_in_bw;
	color_settings_ptr->color_offset = settings.color_offset;

	return OKAY;	
}

	
/*
* Go through the settings list in order of appearance
* Assume that settings_list vector contains data in order of appearence in settings file
* If assumption is false, create error
*/
SettingsLoaderError ExtractSettingsFromVector(std::vector<std::string> settings_list, Settings *settings_ptr){	
	std::stringstream stream;
	std::string settings_key_holder; //A temporary string holder to analyze each part of the retrieved settings file before adding to struct
	int settings_int_value_holder;
	double settings_double_value_holder;
	bool settings_bool_value_holder;

	//Only go through the odd indices as they are the ones that contain the values needed for settings struct

	//Expected: value for dimm
	stream << settings_list[1];
	stream >> settings_int_value_holder;
	if(stream.fail()){
		std::cout << "Invalid settings value for image-dimension" << std::endl;
		return INVALID_SETTINGS_VALUE;
	}
	stream.clear();
	settings_ptr->dimm = settings_int_value_holder;
	
	//Expected: value for radius xy	
	stream << settings_list[3];
	stream >> settings_double_value_holder;
	if(stream.fail()){
		std::cout << "Invalid settings value for image-radius-xy" << std::endl;
		return INVALID_SETTINGS_VALUE;
	}
	stream.clear();
	settings_ptr->radius_xy = settings_double_value_holder;
				
	//Expected: value for x offset
	stream << settings_list[5];
	stream >> settings_double_value_holder;
	if(stream.fail()){
		std::cout << "Invalid settings value for image-graph-x-offset" << std::endl;
		return INVALID_SETTINGS_VALUE;
	}
	stream.clear();
	settings_ptr->x_offset = settings_double_value_holder;
	
	//Expected: value for y offset
	stream << settings_list[7];
	stream >> settings_double_value_holder;
	if(stream.fail()){
		std::cout << "Invalid settings value for image-graph-y-offset" << std::endl;
		return INVALID_SETTINGS_VALUE;
	}
	stream.clear();
	settings_ptr->y_offset = settings_double_value_holder;
	
	//Expected: value for color offset
	stream << settings_list[9];
	stream >> settings_int_value_holder;
	if(stream.fail()){
		std::cout << "Invalid settings value for image-color-offset" << std::endl;
		return INVALID_SETTINGS_VALUE;
	}
	stream.clear();
	settings_ptr->color_offset = settings_int_value_holder;

	//Expected: value for image in color		
	stream << settings_list[11];
	stream >> settings_int_value_holder;
	if(stream.fail()){
		std::cout << "Invalid settings value for image-in-color" << std::endl;
		return INVALID_SETTINGS_VALUE;
	}
	stream.clear();
	if(settings_int_value_holder == 1){
		settings_bool_value_holder = true;
	} else if(settings_int_value_holder == 0){
		settings_bool_value_holder = false;
	} else{
		std::cout << "Invalid settings value for image-in-bw" << std::endl;
		return INVALID_SETTINGS_VALUE;
	}	
	settings_ptr->image_in_bw = settings_bool_value_holder;
	
	//Expected: value for max iterations
	stream << settings_list[13];
	stream >> settings_int_value_holder;
	if(stream.fail()){
		std::cout << "Invalid settings value for max-iterations" << std::endl;
		return INVALID_SETTINGS_VALUE;
	}
	stream.clear();
	settings_ptr->iterations = settings_int_value_holder;

	//Expected: value for escape range
	stream << settings_list[15];
	stream >> settings_int_value_holder;
	if(stream.fail()){
		std::cout << "Invalid settings value for escape-range" << std::endl;
		return INVALID_SETTINGS_VALUE;
	}
	stream.clear();
	settings_ptr->escape_range = settings_int_value_holder;
	
	//Expected: value for fractal type
	stream << settings_list[17];
	stream >> settings_int_value_holder;
	if(stream.fail()){
		std::cout << "Invalid settings value for fractal-type-value" << std::endl;
		return INVALID_SETTINGS_VALUE;
	}
	stream.clear();
	settings_ptr->type = GetFractalTypeFromValue(settings_int_value_holder);


	//Expected: value for constant real
	stream << settings_list[19];
	stream >> settings_double_value_holder;
	if(stream.fail()){
		std::cout << "Invalid settings value for constant-real" << std::endl;
		return INVALID_SETTINGS_VALUE;
	}
	stream.clear();
	settings_ptr->constant_real = settings_double_value_holder;

	//Expected: value for constant imag
	stream << settings_list[21];
	stream >> settings_double_value_holder;
	if(stream.fail()){
		std::cout << "Invalid settings value for constant-imag" << std::endl;
		return INVALID_SETTINGS_VALUE;
	}
	stream.clear();
	settings_ptr->constant_imag = settings_double_value_holder;
	
	return OKAY;

}


} //End namespace
